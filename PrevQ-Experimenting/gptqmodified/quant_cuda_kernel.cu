#include "hip/hip_runtime.h"
// quant_cuda_kernel.cu

#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>


template <typename scalar_t>
__global__ void VecQuant3MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int height,
    int width
);

__global__ void VecQuant3MatMulKernelFaster(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    const  float* __restrict__ zeros,
    int height,
    int width
);

template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const  uint32_t* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const   uint8_t* __restrict__ zeros,
    int height,
    int width
);


const int BLOCKWIDTH  = 256;
const int BLOCKHEIGHT =  24;
const int BLOCKWIDTH_8BIT = 128; // Example block width for 8bit kernel


void vecquant3matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT - 1) / BLOCKHEIGHT,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  AT_DISPATCH_FLOATING_TYPES(
    vec.type(), "vecquant3matmul_cuda", ([&] {
      VecQuant3MatMulKernel<<<blocks, threads>>>(
        vec.data_ptr<scalar_t>(), mat.data_ptr<int>(), mul.data_ptr<scalar_t>(),
        scales.data_ptr<scalar_t>(), zeros.data_ptr<scalar_t>(),
        height, width
      );
    })
  );
}

void vecquant3matmul_faster_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT - 1) / BLOCKHEIGHT,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant3MatMulKernelFaster<<<blocks, threads>>>(
    (half2*) vec.data_ptr(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    scales.data_ptr<float>(),
    zeros.data_ptr<float>(),
    height, width
  );
}


void vecquant8matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales,
  torch::Tensor zeros
) {
  // mat shape: [infeatures / 4, outfeatures] -> height = infeatures / 4, width = outfeatures
  // vec shape: [infeatures]
  // mul shape: [outfeatures]
  // scales shape: [outfeatures]
  // zeros shape: [outfeatures] (uint8)
  int packed_rows = mat.size(0);
  int width = mat.size(1); // outfeatures
  int height = vec.size(0); // infeatures

  // Launch configuration maps threads to output columns (width)
  dim3 blocks((width + BLOCKWIDTH_8BIT - 1) / BLOCKWIDTH_8BIT);
  dim3 threads(BLOCKWIDTH_8BIT);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    vec.scalar_type(), "vecquant8matmul_cuda", ([&] {
      VecQuant8MatMulKernel<<<blocks, threads>>>(
        vec.data_ptr<scalar_t>(),
        mat.data_ptr<uint32_t>(),
        mul.data_ptr<scalar_t>(),
        scales.data_ptr<scalar_t>(),
        zeros.data_ptr<uint8_t>(),
        height, // infeatures
        width   // outfeatures
      );
    })
  );
}


__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}


template <typename scalar_t>
__global__ void VecQuant3MatMulKernel(
    const  scalar_t* __restrict__ vec,
    const       int* __restrict__ mat,
           scalar_t* __restrict__ mul,
    const  scalar_t* __restrict__ scales,
    const  scalar_t* __restrict__ zeros,
    int height,
    int width
) {
  int row = BLOCKHEIGHT * blockIdx.x;
  int col =  BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ scalar_t blockvec[BLOCKWIDTH];
  if (threadIdx.x < BLOCKWIDTH) {
      int vec_idx = (row / BLOCKHEIGHT) * BLOCKWIDTH + threadIdx.x;
       if (vec_idx < height) { // Ensure reading within bounds of vec (height=infeatures)
          blockvec[threadIdx.x] = vec[vec_idx];
       } else {
           blockvec[threadIdx.x] = 0; // Pad with zero if out of bounds
       }
  }
  __syncthreads();

  if (col >= width) return; // Check column bounds

  scalar_t scale = scales[col];
  scalar_t zero = zeros[col]; // Float zero point for 3bit

  scalar_t res = 0;
  int i = width * row + col; // Index into mat [height=rows(packed), width=cols(outfeatures)]
  int k = 0;

  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;

  // Assuming height is multiple of BLOCKHEIGHT for simplicity in loop structure
  // A more robust kernel handles arbitrary height/infeatures
  while (k < BLOCKWIDTH) {
    tmp1 = as_unsigned(mat[i]);
    res += (scale * scalar_t((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width; // Move to the next packed row for this column
    tmp2 = as_unsigned(mat[i]);
    tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x4);
    tmp2 >>= 1;
    res += (scale * scalar_t(tmp) - zero) * blockvec[k + 10];
    k += 11;
    res += (scale * scalar_t((tmp2 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp2 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp2 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp2 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp2 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp2 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp2 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp2 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp2 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp2 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    tmp1 = as_unsigned(mat[i]);
    tmp = (tmp2 >> 30) | ((tmp1 << 1) & 0x6);
    tmp1 >>= 2;
    res += (scale * scalar_t(tmp) - zero) * blockvec[k + 10];
    k += 11;
    res += (scale * scalar_t((tmp1 >>  0) & 0x7) - zero) * blockvec[k + 0];
    res += (scale * scalar_t((tmp1 >>  3) & 0x7) - zero) * blockvec[k + 1];
    res += (scale * scalar_t((tmp1 >>  6) & 0x7) - zero) * blockvec[k + 2];
    res += (scale * scalar_t((tmp1 >>  9) & 0x7) - zero) * blockvec[k + 3];
    res += (scale * scalar_t((tmp1 >> 12) & 0x7) - zero) * blockvec[k + 4];
    res += (scale * scalar_t((tmp1 >> 15) & 0x7) - zero) * blockvec[k + 5];
    res += (scale * scalar_t((tmp1 >> 18) & 0x7) - zero) * blockvec[k + 6];
    res += (scale * scalar_t((tmp1 >> 21) & 0x7) - zero) * blockvec[k + 7];
    res += (scale * scalar_t((tmp1 >> 24) & 0x7) - zero) * blockvec[k + 8];
    res += (scale * scalar_t((tmp1 >> 27) & 0x7) - zero) * blockvec[k + 9];
    i += width;
    k += 10;
  }

  atomicAdd(&mul[col], res);
}

__global__ void VecQuant3MatMulKernelFaster(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    const  float* __restrict__ zeros,
    int height,
    int width
) {
  const int blockwidth2 = BLOCKWIDTH / 2;

  int row = BLOCKHEIGHT * blockIdx.x;
  int col = BLOCKWIDTH * blockIdx.y + threadIdx.x;

   __shared__ half2 blockvec[blockwidth2];
   if (threadIdx.x < blockwidth2) {
       int vec_idx = (row / BLOCKHEIGHT) * blockwidth2 + threadIdx.x;
        // Assuming vec has length height = infeatures
       if (vec_idx * 2 + 1 < height) { // Check bounds for half2
           blockvec[threadIdx.x] = vec[vec_idx];
       } else if (vec_idx * 2 < height) {
           // Handle case where only the first half is valid
           // This requires careful handling or assuming vec length is even
            blockvec[threadIdx.x] = __halves2half2(((half*)vec)[vec_idx * 2], __float2half(0.0f));
       }
       else {
            blockvec[threadIdx.x] = __float2half2_rn(0.0f); // Pad with zero
       }
   }

  __shared__ half2 deq2[64][32];
  int val = threadIdx.x / 32;
  int off = threadIdx.x % 32;
  for (; val < 64; val += BLOCKWIDTH / 32) {
    deq2[val][off] = __halves2half2(
       __int2half_rn(val & 0x7), __int2half_rn(val >> 3)
    );
  }

  __syncthreads();

  if (col >= width) return; // Check column bounds

  half2 scale = __float2half2_rn(scales[col]);
  half2 zero = __float2half2_rn(-zeros[col]); // Note: using -zeros[col]

  int i = width * row + col;
  int k = 0;

  float res = 0;
  half2 res2;

  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;


  while (k < blockwidth2) { // Iterate through input features (paired in half2)
    res2 = {};
    tmp1 = as_unsigned(mat[i]);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 24) & 0x3f][off], scale, zero), blockvec[k + 4], res2);
    i += width;
    tmp2 = as_unsigned(mat[i]);
    tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x3c);
    res2 = __hfma2(__hfma2(deq2[tmp][off], scale, zero), blockvec[k + 5], res2);
    tmp2 >>= 4;
    k += 6;
    res2 = __hfma2(__hfma2(deq2[(tmp2 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp2 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp2 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp2 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
    i += width;
    tmp1 = as_unsigned(mat[i]);
    tmp = (tmp2 >> 24) | ((tmp1 << 4) & 0x30);
    res2 = __hfma2(__hfma2(deq2[tmp][off], scale, zero), blockvec[k + 4], res2);
    tmp1 >>= 2;
    k += 5;
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 24) & 0x3f][off], scale, zero), blockvec[k + 4], res2);
    i += width;
    k += 5;
    res += __half2float(res2.x) + __half2float(res2.y);
  }

  atomicAdd(&mul[col], res);
}


template <typename scalar_t>
__global__ void VecQuant8MatMulKernel(
    const  scalar_t* __restrict__ vec,     // Input vector [infeatures]
    const  uint32_t* __restrict__ mat,     // Packed weights [infeatures / 4, outfeatures]
           scalar_t* __restrict__ mul,     // Output vector [outfeatures]
    const  scalar_t* __restrict__ scales,  // Scales [outfeatures]
    const   uint8_t* __restrict__ zeros,   // Integer zero points [outfeatures]
    int height,                           // Infeatures dimension
    int width                             // Outfeatures dimension
) {
    int out_col = blockIdx.x * blockDim.x + threadIdx.x; // Maps threads to output columns

    if (out_col >= width) return; // Ensure thread is within output bounds

    scalar_t scale = scales[out_col];
    uint8_t zero_point = zeros[out_col];
    scalar_t zero_point_f = static_cast<scalar_t>(zero_point);

    scalar_t accum = 0;

    // Iterate over the input features (height = infeatures)
    for (int k = 0; k < height; ++k) {
        // Calculate index into packed matrix
        int packed_row = k / 4;
        int sub_idx = k % 4; // Which 8-bit weight within the uint32 (0, 1, 2, or 3)

        // Read the packed uint32 value
        // Index: packed_row * width + out_col (Column-major access)
        uint32_t packed_val = mat[packed_row * width + out_col];

        // Unpack the specific 8-bit weight
        uint8_t q_val = (packed_val >> (sub_idx * 8)) & 0xFF;

        // Dequantize: W = scale * (Q - zero_point)
        scalar_t w_val = scale * (static_cast<scalar_t>(q_val) - zero_point_f);

        // Multiply and accumulate
        accum += w_val * vec[k];
    }

    // Atomically add the result for this output column
    // Assumes mul is initialized to zero before kernel launch
    atomicAdd(&mul[out_col], accum);
}